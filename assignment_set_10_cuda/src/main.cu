#include "hip/hip_runtime.h"
#include "../lib/sph_kernel.cuh"
#include <fstream>

//SPH toy Star with gravity using CUDA
void init_particles(ParticleData& particles, int N) {
    hipMalloc(&particles.pos, N * sizeof(double3));
    hipMalloc(&particles.vel, N * sizeof(double3));
    hipMalloc(&particles.acc, N * sizeof(double3));
    hipMalloc(&particles.linear_acc_fore, N * sizeof(double3));
    hipMalloc(&particles.damping_force, N * sizeof(double3));

    hipMalloc(&particles.mass, N * sizeof(double));
    hipMalloc(&particles.rho, N * sizeof(double));
    hipMalloc(&particles.pressure, N * sizeof(double));
    hipMalloc(&particles.cs, N * sizeof(double));
}

void free_particles(ParticleData& particles) {
    hipFree(particles.pos);
    hipFree(particles.vel);
    hipFree(particles.acc);
    hipFree(particles.linear_acc_fore);
    hipFree(particles.damping_force);

    hipFree(particles.mass);
    hipFree(particles.rho);
    hipFree(particles.pressure);
    hipFree(particles.cs);
}

int main(){
    ParticleData particles;
    hipMalloc(&particles.pos, NUM_PARTICLES * sizeof(double3));
    hipMalloc(&particles.vel, NUM_PARTICLES * sizeof(double3));
    hipMalloc(&particles.acc, NUM_PARTICLES * sizeof(double3));
    hipMalloc(&particles.pressure, NUM_PARTICLES * sizeof(double));
    hipMalloc(&particles.rho, NUM_PARTICLES * sizeof(double));
    hipMalloc(&particles.cs, NUM_PARTICLES * sizeof(double));
    hipMalloc(&particles.linear_acc_fore, NUM_PARTICLES * sizeof(double3));
    hipMalloc(&particles.damping_force, NUM_PARTICLES * sizeof(double3));
    // Initialize host data
    std::vector<double3> h_pos(NUM_PARTICLES);
    std::vector<double3> h_vel(NUM_PARTICLES);
    std::vector<double3> h_acc(NUM_PARTICLES);
    std::vector<double> h_pressure(NUM_PARTICLES);
    std::vector<double> h_rho(NUM_PARTICLES);
    std::vector<double> h_cs(NUM_PARTICLES);
    std::vector<double> h_mass(NUM_PARTICLES);
    std::vector<double3> h_linear_acc_fore(NUM_PARTICLES);
    std::vector<double3> h_damping_force(NUM_PARTICLES);
    // Initialize host data with distribution from file
    std::ifstream infile("../initial_dis/random_distribution.dat");
    if(!infile) {
        std::cerr << "Error opening initial data file\n";
        return 1;
    }
    for(int i = 0; i < NUM_PARTICLES; ++i) {
        infile >> h_pos[i].x >> h_pos[i].y >> h_pos[i].z >> h_vel[i].x >> h_vel[i].y >> h_vel[i].z >> h_mass[i];
        h_acc[i] = {0.0, 0.0, 0.0}; // Initial acceleration
        h_pressure[i] = 0.0; // Initial pressure
        h_rho[i] = 1.0; // Initial density
        h_cs[i] = 1.0; // Initial sound speed
        h_linear_acc_fore[i] = {0.0, 0.0, 0.0}; // Initial linear acceleration
        h_damping_force[i] = {0.0, 0.0, 0.0}; // Initial damping force
    }


}