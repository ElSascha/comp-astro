#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(double *a, double *b, double *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

int main(){
    const int N = 10;
    double h_a[N], h_b[N], h_c[N];

    // Init Host Arrays
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = 100 + i;
    }

    // Device Pointer
    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(double));
    hipMalloc(&d_b, N * sizeof(double));
    hipMalloc(&d_c, N * sizeof(double));

    // Kopieren: Host → Device
    hipMemcpy(d_a, h_a, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(double), hipMemcpyHostToDevice);

    // CUDA-Kernel starten
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    add<<<blocks, threadsPerBlock>>>(d_a, d_b, d_c);

    // Ergebnis zurück: Device → Host
    hipMemcpy(h_c, d_c, N * sizeof(double), hipMemcpyDeviceToHost);

    // Ausgabe
    for (int i = 0; i < N; ++i)
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << "\n";

    // Aufräumen
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
    
}